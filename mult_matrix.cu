#include "hip/hip_runtime.h"
#include "multShare.h"

void MatMul(const Matrix A, const Matrix B, Matrix C) {

  Matrix d_A;
  d_A.width = d_A.stride = A.width;
  d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipError_t err = hipMalloc(&d_A.elements, size);
  printf("CUDA malloc A: %s\n",hipGetErrorString(err));
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width = d_B.stride = B.width;
  d_B.height = B.height;
  size = B.width * B.height * sizeof(float);
  err = hipMalloc(&d_B.elements, size);
  printf("CUDA malloc B: %s\n",hipGetErrorString(err));
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
  Matrix d_C;
  d_C.width = d_C.stride = C.width;
  d_C.height = C.height;
  size = C.width * C.height * sizeof(float);
  err = hipMalloc(&d_C.elements, size);
  printf("CUDA malloc C: %s\n",hipGetErrorString(err));
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    err = hipDeviceSynchronize();
    printf("Run kernel: %s\n", hipGetErrorString(err));
  err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
  printf("Copy C off of device: %s\n",hipGetErrorString(err));

  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}


__device__ float GetElement(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
}

__device__ void SetElement(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
}

__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
  Matrix Asub;
  Asub.width = BLOCK_SIZE;
  Asub.height = BLOCK_SIZE;
  Asub.stride = A.stride;
  Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
  return Asub;
}

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {

  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;

  Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

  float Cvalue = 0.0;
  int row = threadIdx.y;
  int col = threadIdx.x;
  
  for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
   
    Matrix Asub = GetSubMatrix(A, blockRow, m);
    Matrix Bsub = GetSubMatrix(B, m, blockCol);

    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    As[row][col] = GetElement(Asub, row, col);
    Bs[row][col] = GetElement(Bsub, row, col);

    __syncthreads();

    for (int e = 0; e < BLOCK_SIZE; ++e)
      Cvalue += As[row][e] * Bs[e][col];

    __syncthreads();
  }
  SetElement(Csub, row, col, Cvalue);
}


int main(int argc, char* argv[]){
    Matrix A, B, C;
    int a1, a2, b1, b2;
    a1 = atoi(argv[1]); 
    a2 = atoi(argv[2]); 
    b1 = a2;     
    b2 = atoi(argv[3]); 
    A.height = a1;
    A.width = a2;
    A.elements = (float*)malloc(A.width * A.height * sizeof(float));
    B.height = b1;
    B.width = b2;
    B.elements = (float*)malloc(B.width * B.height * sizeof(float));
    C.height = A.height;
    C.width = B.width;
    C.elements = (float*)malloc(C.width * C.height * sizeof(float));
    for(int i = 0; i < A.height; i++) {
      for(int j = 0; j < A.width; j++) {
        A.elements[i*A.width + j] = (random() % 3);
      }
    }    
    for(int i = 0; i < B.height; i++) {
      for(int j = 0; j < B.width; j++) {
        B.elements[i*B.width + j] = (random() % 2);
      }
    }

    MatMul(A, B, C);

    for(int i = 0; i < min(10, A.height); i++) {
      for(int j = 0; j < min(10, A.width); j++) {
        printf("%f ", A.elements[i*A.width + j]);
      }
      printf("\n");
    }
    printf("\n");
    for(int i = 0; i < min(10, B.height); i++){
      for(int j = 0; j < min(10, B.width); j++){
        printf("%f ", B.elements[i*B.width + j]);
      }
      printf("\n");
    }
    printf("\n");  
    for(int i = 0; i < min(10, C.height); i++){
      for(int j = 0; j < min(10, C.width); j++){
        printf("%f ", C.elements[i*C.width + j]);
      }
      printf("\n");
    }
    printf("\n");
}    